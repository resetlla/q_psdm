#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <errno.h>
#include <math.h>
#include <pthread.h>
#include <stdint.h>
#include <sys/time.h>
#include <time.h>
#include "fft.h"
#include "psdmpkg.h"
#include "queue.h"
#include "segy.h"

#include <sstream>
#include <string>
#include <vector>

#define QMIG3D
#define FULL_DERIVATIVE

// #define  WARN_ON

#ifdef WARN_ON
#define WARN(fmt, ...)                   \
  do {                                   \
    fprintf(stderr, fmt, ##__VA_ARGS__); \
  } while (0)
#else
#define WARN(fmt, ...)
#endif

const char *sdoc[] = {
#ifdef DMIG3D_WITHOUT_GATHER
    "Prestack Depth Migration: Dip Migration With Multithreading             "
    "\n",
#else
    "Prestack Depth Migration: Generate Dip-Angle Gather With Multithreading "
    "\n",
#endif
    "                          Using grid travel time table                  "
    "\n",
    " dgthr3dz\033[1;31m path=/..  parfilepath=.. nodename=**_[0|1..]         "
    "\n",
    "         ttpath=/..                                      \033[0m        "
    "\n",
    "        *[offspath=/..] *[noffpath=/..] *[velfilepath=/..]              "
    "\n",
    "        *[infopath=/..] *[dipdirpath=/..] *[outputpath=/..]             "
    "\n",
    "        *[logfilepath=/..] *[traveltime=2|4|6] *[ninv=20]               "
    "\n",
    "        *[f1=1] *[f2=3] *[f3=130.0] *[f4=140](Hz)                       "
    "\n",
    "        *fxs= dxs= nxs= fys= dys= nys=                                  "
    "\n",
    "        *dxt= nxt= dyt= nyt= dzt= nzt=                                  "
    "\n",
#ifdef DMIG3D_WITHOUT_GATHER
    "        *[opcrp=1] *[cdps=#cdp1-#cdp2] *[aspect=N]                      "
    "\n",
#else
    "        *[opcrp=0] *[cdps=#cdp1-#cdp2] *[aspect=A|X|Y|N]                "
    "\n",
    "        *[dipx=50] *[dipy=50] *[ddipx=1] *[ddipy=1](degree)             "
    "\n",
#endif
    "        *dyv=#header                                                  "
    "\n\n",
    "        curl=NULL                                                       "
    "\n",
    "    traveltime: 2-DSR 4-TANER2TH 6-TANER4THCUT                          "
    "\n",
    "          ninv: Time Ratio Used to Inverse the Interval Velocity        "
    "\n",
    "    f1,2,3,4  : Filter Parameter Applied to the Input Seismic Data      "
    "\n",
#ifdef DMIG3D_WITHOUT_GATHER
    "                                                                      "
    "\n\n",
#else
    "    opcrp     : 0, not output CRP                                       "
    "\n",
    "                1, output CRP                                           "
    "\n",
    "    cdps      : specify the CDPs output in CRP                          "
    "\n",
    "    aspect    : All,X+Y;  X; Y; N  for Dip-Gather                       "
    "\n",
    "    dipx,dipy : Maximum dip for X,Y Direction                           "
    "\n",
    "    ddipx,y   : Interval                                              "
    "\n\n",
#endif
    "\033[1;34m  SU Format     \033[0m                                     "
    "\n\n",
    " * indicate the path is derivate from the {path}                      "
    "\n\n",
    NULL};

segy tr, tr1;
FILE *fplog;
int is3D = 1;

typedef struct {
  int64_t idx;
  uint16_t nys;
  uint16_t nxs;
  uint16_t dys;
  uint16_t dxs;
} ttt_desc_t;

struct CPUParamsNoLine {
  // 从main中读取的参数
  float dyi;
  float dxi;
  int nxi;
  int nzi;
  int ntl;
  double scaled;
  int fci;
  float dzi;
  int ndipx;
  int ndipy;
  float ddipx;
  float ddipy;
  float dt;
  float fc;
  // 内部参数
  float eps;
  int *iaztab;
  float fxv1;
  float fyv1;
  float exv1;
  float eyv1;
  int nfft;
  int nffti;
  float *ww;
  float dt1;
  int ndipx2;
  int ndipy2;
  float lnG;
  float coef1;
  float coef2;
  float coef3;
  float coef4;
  float taperzone;
  float dzii;
};

struct CPUParamsWithLine {
  double fxi;
  double cyi;
  float dw;
  int ****aper;
  int **aznx1;
  int **aznx2;
  int **azny1;
  int **azny2;
  int ioffs;
  int itb1;
  int itb2;
  int nsb;
  float tdstart1;
  float tdstart2;
  int nf1;
  float tdmid;
  int nww;
  int nxg;
  int nzg;
  int nxi1;
  int nzi1;
  int ttt_cdp1;
  int ttt_cdp2;
  int qqq_cdp1;
  int qqq_cdp2;
  int ncoef1;
  int ixstart;
  int ixend;
  int nxb;
};

struct GPUParamsNoLine {
  int igpu;
  float *re;
  float *im;
  float *red;
  float *imd;
  int *itibeg;
  int *itibegd;
};

struct GPUParamsWithLine {
  int *nf3d;
  float *Qd;
  float *datav;
  fftwf_complex *wdatav;
  fftwf_plan planv;
  ttt_desc_t *qqq_descd;
  float *qqqd;
  fftwf_plan plan;
  fftwf_plan plani;
  fftwf_complex *wdata;
  float *data;
  float *datad;
  float *dipx1d;
  float *dipx2d;
  float *dipy1d;
  float *dipy2d;
  ttt_desc_t *ttt_descd;
  float *tttd;
  float *imgd;
  float *gthrxd;
  float *gthryd;
};

struct PROD_PARAMS {
  safe_queue<segy *> *datapool;
  int ioffl1;
  int ioffl2;
  char *offspath;
  int ioffs;
  int ntl;
  int ncons;
  char *nodename;
  uint64_t *t_read;
  off_t *s_read;
};

struct CONS_PARAMS {
  safe_queue<segy *> *datapool;
  CPUParamsNoLine *cpuParamsNoLine;
  CPUParamsWithLine *cpuParamsWithLine;
  GPUParamsNoLine *gpuParamsNoLine;
  GPUParamsWithLine *gpuParamsWithLine;
};

void *producer(void *arg);
void *consumer(void *arg);

void qpsdm(int ngpu, int ndata, int ncons, int poolsize, char *projpath,
           char *projtask, float dxi, int nxi, float dyi, int nyi, int nzi,
           float dzi, float dzii, float zmin, int fcv, double fxv, int flv,
           double fyv, int nxv, int nyv, int ntv, float dtv, int naxi, int nayi,
           float daxi, float dayi, int line1, int line2, int nyd, int fld,
           int ntl, float dt, float f1, float fc, float f3, float f4,
           int *noffs, int extxline, int *iline, int fci, int iyi1,
           float **offs, int *cdps, int ncdps, int nsmooth, double scaled,
           float **tmute, char *infopath, int aspect, int opcrp,
           char *outputpath, char *offspath, char *path, char *velfilepath,
           char *dipdirpath, char *f3dirpath, char *ttpath, char *nodename,
           float threshold, float contract, float taperzone, int nblock);

void psdm_kernel(segy tr,
                 // CPUParamsNoLine
                 float dyi, float dxi, int nxi, int nzi, int ntl, double scaled,
                 int fci, float dzi, int ndipx, int ndipy, float ddipx,
                 float ddipy, float dt, float fc, float eps, int *iaztab,
                 float fxv1, float fyv1, float exv1, float eyv1, int nfft,
                 int nffti, float *ww, float dt1, int ndipx2, int ndipy2,
                 float taperzone, float lnG, float coef1, float coef2,
                 float coef3, float coef4, float dzii,
                 // CPUParamsWithLine
                 int ****aper, int **aznx1, int **aznx2, int **azny1,
                 int **azny2, double fxi, double cyi, float dw, int ioffs,
#ifdef QMIG3D
                 int itb1, int itb2, int nsb, float tdstart1, float tdstart2,
                 int nf1, float tdmid,
#endif
                 int nww, int nxg, int nzg, int nxi1, int nzi1, int ttt_cdp1,
                 int ttt_cdp2, int qqq_cdp1, int qqq_cdp2, int ncoef1,
                 int ixstart, int ixend, int nxb,
// GPUParamsNoLine
#ifdef QMIG3D
                 float *re, float *im, float *red, float *imd,
#endif
                 int igpu, int *itibeg, int *itibegd,
// GPUParamsWithLine
#ifdef QMIG3D
                 int *nf3d, float *Qd, float *datav, fftwf_complex *wdatav,
                 fftwf_plan planv, ttt_desc_t *qqq_descd, float *qqqd,
#endif
                 float *data, float *datad, float *dipx1d, float *dipx2d,
                 float *dipy1d, float *dipy2d, ttt_desc_t *ttt_descd,
                 float *tttd, float *imgd);

__global__ void image_depth_gpu(
    float *img, int izimin, int nzi, float *data, int nt, float dt, int ixstart,
    int ixend, int nxb, float dzii,
#ifdef QMIG3D
    const float *__restrict__ red, const float *__restrict__ imd, int nf1,
    float fc, float dw, int nww, int *nf3, ttt_desc_t *qqq_desc, float *qqq,
    float tstart1, float tstart2, float tmid,
#endif
    float taperzone, float lnG, float coef1, float coef2, float coef3,
    float coef4, int ncoef1, int nxg, int nzg, int nxi1, int nzi1, int ttt_cdp1,
    int ttt_cdp2, float sx, float sy, float gx, float gy, float dxi, float dzi,
    int *itibegd, int ixi1, int ixi2, int ndipx, int ndipx2, float ddipx,
    int ndipy, int ndipy2, float ddipy, ttt_desc_t *ttt_desc, float *ttt,
    float *dipx1, float *dipx2, float *dipy1, float *dipy2);

int get_ttt_all(const char *ttpath, const char *torq, int line, int line1,
                int line2, int *nxg, int *nzg, int *nxi1, int *nzi1,
                int *ttt_cdp1, int *ttt_cdp2,
                std::vector<GPUParamsWithLine *> gpuParamsWithLineVec,
                int ncons);
void reset_ttt_all(ttt_desc_t *desc_out, float *data_out);

int main(int argc, char *argv[]) {
  /* 命令行参数 */
  char *path, *ttpath, *parfilepath, *nodename, *offspath, *noffpath,
      *velfilepath, *dipdirpath, *f3dirpath, *outputpath, *logfilepath,
      *infopath, *zcdps, *zaspect;
  int ninv;
  float f1, f2, f3, f4, fc;
  float threshold, contract, taperzone;

  /* 参数文件参数:数据空间 */
  int ntl, line1, line2; /* Time Sample Number,First Data Line,Last Data Line */
  float dt;              /* Time Sample Interval                              */

  /* 参数文件参数:成像空间 */
  int cdp1, cdp2; /* Image Range:cdp,lines,time */
  float dzi, zmin, zmax, dzii;
  int *iline, nyi;
  char *zline;

  /* 参数文件参数:切除和孔径 */
  char *zmute;
  int xline;

  /* 速度空间变量 */
  double fxv, fyv;
  float dxv, dyv, dtv, dzv;
  int flv, fcv, nxv, nyv, ntv;

  /* 成像空间变量 */
  int fci, nxi, nzi;
  /* AUX */
  float dyi, dxi;

  /* 数据空间变量 */
  int fld, nyd;

  /* 偏移距 */
  int *noffs;
  float offmin, offmax;
  float **offs = NULL;

  /* 切除信息 */
  float **tmute = NULL;

  /* 检查点信息变量 */
  int iyi1, ilb;

  /* 常规索引 */
  int idata, ioffs;

  /* MISC */
  char str[512];
  double scalev, scaled;
  int ret, nsmooth;
  int igpu, ngpu;
  FILE *fp;
  int i, line;
  time_t tic, toc;

  /* 命令行拼写参数 */
  int iproc;
  char projpath[1024];
  char projname[512];
  char projtask[512];

  /* 切除曲线 */
  float *tomute;
  int ntomute, npair;

  hipError_t err;

  /* 倾角道集相关 */
  int opcrp, aspect;
  float dipx, dipy, ddipx, ddipy;
  int *cdps, ncdps;
  int ndipx, ndipy;

  /*	分块 */
  int nblock;

  /* 获取命令行参数:强制 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  initargs(argc, argv, sdoc);

  if (!getparstring("path", &path)) {
    fprintf(stderr, "Error: path must be specified\n");
    exit(1);
  }
  /* 去掉path最后的一个或多个/(如果存在的话) */
  strrtrim(path, '/');

  if (!getparstring("parfilepath", &parfilepath)) {
    fprintf(stderr, "Error: parfilepath must be specified\n");
    exit(1);
  }

  if (!getparstring("nodename", &nodename)) {
    fprintf(stderr, "Error: nodename must be specified\n");
    exit(1);
  }

  if (!getparstring("ttpath", &ttpath)) {
    fprintf(stderr, "Error: ttpath must be specified\n");
    exit(1);
  }

  /*	多线程参数 */
  int ndata, ncons, poolsize;

  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

  /* 解析path到各个子项 */
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  parsepath(path, projpath, projname, &iproc, projtask);
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

  /* 再次获取命令行参数:弱获取 */
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  if (!getparstring("logfilepath", &logfilepath)) {
    logfilepath = alloc1char(1024);
    if (1 != readparfile(parfilepath, "logfilepath=%s", logfilepath)) {
      sprintf(logfilepath, "/tmp/%s_%s.log", nodename, projname);
    }
  }

  /* 日志系统开启 */
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  fplog = stderr;  // fopen(logfilepath,"w");
  if (fplog == NULL) {
    fprintf(stderr, "Warn: log file [%s] open error\n", logfilepath);
    fprintf(stderr, "      redirect it to stderr\n");
    fplog = stderr;
  }
  setbuf(fplog, NULL);

  time(&tic);
  fprintf(fplog,
          "++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++"
          "++\n");
  fprintf(fplog,
          "++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++"
          "++\n");
  fprintf(fplog,
          "+ Log File For Prestack Time Migration, Generate Dip Gather With "
          "Multithreading @ %s \n",
          nodename);
  fprintf(fplog,
          "++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++"
          "++\n");
  fprintf(fplog,
          "++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++"
          "++\n");
  fprintf(fplog, "Time: %s\n", ctime(&tic));
  fflush(fplog);
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

  if (!getparstring("offspath", &offspath)) {
    offspath = alloc1char(1024);
    if (1 != readparfile(parfilepath, "offspath=%s", offspath)) {
      sprintf(offspath, "%s/data%d", projpath, iproc);
    }
  }

  if (!getparstring("velfilepath", &velfilepath)) {
    velfilepath = alloc1char(1024);
    if (1 != readparfile(parfilepath, "velfilepath=%s", velfilepath)) {
      sprintf(velfilepath, "%s/vel/vel_loadin", projpath);
    }
  }

  if (!getparstring("infopath", &infopath)) {
    infopath = alloc1char(1024);
    if (1 != readparfile(parfilepath, "infopath=%s", infopath)) {
      strcpy(infopath, path);
    }
  }

  if (!getparstring("outputpath", &outputpath)) {
    outputpath = alloc1char(1024);
    if (1 != readparfile(parfilepath, "outputpath=%s", outputpath)) {
      sprintf(outputpath, "%s/result", path);
    }
  }

  if (!getparint("ninv", &ninv)) {
    if (1 != readparfile(parfilepath, "ninv=%d", &ninv)) {
      ninv = 20;
    }
  }

  if (!getparfloat("f1", &f1)) {
    if (1 != readparfile(parfilepath, "f1=%f", &f1)) {
      f1 = 1;
    }
  }

  if (!getparfloat("fc", &fc)) {
    if (1 != readparfile(parfilepath, "fc=%f", &fc)) {
      fc = 40;
    }
  }

  if (!getparfloat("contract", &contract)) {
    if (1 != readparfile(parfilepath, "contract=%f", &contract)) {
      contract = 3.0;
    }
  }

  if (!getparfloat("taperzone", &taperzone)) {
    if (1 != readparfile(parfilepath, "taperzone=%f", &taperzone)) {
      taperzone = 8.0;
    }
  }

  if (!getparfloat("threshold", &threshold)) {
    if (1 != readparfile(parfilepath, "threshold=%f", &threshold)) {
      threshold = 600.0;
    }
  }

  if (!getparfloat("f2", &f2)) {
    if (1 != readparfile(parfilepath, "f2=%f", &f2)) {
      f2 = 4;
    }
  }
  if (!getparfloat("f3", &f3)) {
    if (1 != readparfile(parfilepath, "f3=%f", &f3)) {
      f3 = 130;
    }
  }

  if (!getparfloat("f4", &f4)) {
    if (1 != readparfile(parfilepath, "f4=%f", &f4)) {
      f4 = 140;
    }
  }

  if (!getparint("ntl", &ntl)) {
    if (1 != readparfile(parfilepath, "ntl=%d", &ntl)) {
      fprintf(stderr, "Error: ntl must be specified\n");
      fprintf(fplog, "Error: ntl must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparfloat("dt", &dt)) {
    if (1 != readparfile(parfilepath, "dt=%f", &dt)) {
      fprintf(stderr, "Error: dt must be specified\n");
      fprintf(fplog, "Error: dt must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparint("line1", &line1)) {
    if (1 != readparfile(parfilepath, "line1=%d", &line1)) {
      fprintf(stderr, "Error: line1 must be specified\n");
      fprintf(fplog, "Error: line1 must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparint("line2", &line2)) {
    if (1 != readparfile(parfilepath, "line2=%d", &line2)) {
      fprintf(stderr, "Error: line2 must be specified\n");
      fprintf(fplog, "Error: line2 must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparfloat("zmin", &zmin)) {
    if (1 != readparfile(parfilepath, "zmin=%f", &zmin)) {
      fprintf(stderr, "Error: zmin must be specified\n");
      fprintf(fplog, "Error: zmin must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparfloat("zmax", &zmax)) {
    if (1 != readparfile(parfilepath, "zmax=%f", &zmax)) {
      fprintf(stderr, "Error: zmax must be specified\n");
      fprintf(fplog, "Error: zmax must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparfloat("dzi", &dzi)) {
    if (1 != readparfile(parfilepath, "dzi=%f", &dzi)) {
      fprintf(stderr, "Error: dzi must be specified\n");
      fprintf(fplog, "Error: dzi must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparint("cdp1", &cdp1)) {
    if (1 != readparfile(parfilepath, "cdp1=%d", &cdp1)) {
      fprintf(stderr, "Error: cdp1 must be specified\n");
      fprintf(fplog, "Error: cdp1 must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparint("cdp2", &cdp2)) {
    if (1 != readparfile(parfilepath, "cdp2=%d", &cdp2)) {
      fprintf(stderr, "Error: cdp2 must be specified\n");
      fprintf(fplog, "Error: cdp2 must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparstring("line", &zline)) {
    zline = alloc1char(2048);
    if (1 != readparfile(parfilepath, "line=%s", zline)) {
      fprintf(stderr, "Error: line must be specified\n");
      fprintf(fplog, "Error: line must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  if (!getparstring("mute", &zmute)) {
    zmute = alloc1char(2048);
    if (1 != readparfile(parfilepath, "mute=%s", zmute)) {
      strcpy(zmute, "(1,9999),(100,9999)");
    }
  }

  if (!getparint("extxline", &xline)) {
    if (1 != readparfile(parfilepath, "extxline=%d", &xline)) {
      fprintf(stderr, "Error: extxline must be specified\n");
      fprintf(fplog, "Error: extxline must be specified\n");
      fflush(fplog);
      exit(1);
    }
  }

  /* Dip Gather Related */
  if (!getparint("opcrp", &opcrp)) {
    if (1 != readparfile(parfilepath, "opcrp=%d", &opcrp)) {
#ifdef DMIG3D_WITHOUT_GATHER
      opcrp = 1;
#else
      opcrp = 0;
#endif
    }
  }

  if (!getparstring("cdps", &zcdps)) {
    zcdps = alloc1char(1024);
    if (1 != readparfile(parfilepath, "cdps=%s", zcdps)) {
      sprintf(zcdps, "%d-%d", cdp1, cdp2);
    }
  }

  if (!getparstring("aspect", &zaspect)) {
    zaspect = alloc1char(16);
    if (1 != readparfile(parfilepath, "aspect=%s", zaspect)) {
#ifdef DMIG3D_WITHOUT_GATHER
      strcpy(zaspect, "None");
#else
      strcpy(zaspect, "All");
#endif
    }
  }

  if (!getparfloat("dipx", &dipx)) {
    if (1 != readparfile(parfilepath, "dipx=%f", &dipx)) {
      dipx = 50.0;
    }
  }

  if (!getparfloat("dipy", &dipy)) {
    if (1 != readparfile(parfilepath, "dipy=%f", &dipy)) {
      dipy = 50.0;
    }
  }

  if (!getparfloat("ddipx", &ddipx)) {
    if (1 != readparfile(parfilepath, "ddipx=%f", &ddipx)) {
      ddipx = 1.0;
    }
  }

  if (!getparfloat("ddipy", &ddipy)) {
    if (1 != readparfile(parfilepath, "ddipy=%f", &ddipy)) {
      ddipy = 1.0;
    }
  }

  if (!getparint("nblock", &nblock)) {
    if (1 != readparfile(parfilepath, "nblock=%d", &nblock)) {
      nblock = 1;
    }
  }

  /* 部分合理性检测 */
  if (f3 > 0.5 / dt - 10.0) /* freq(nyq)-10 hz */
  {
    f3 = 0.5 / dt - 10.0;
  }

  if (f4 > 0.5 / dt - 5.0) /* freq(nyq)-10 hz */
  {
    f4 = 0.5 / dt - 5.0;
  }

  if (f4 - f3 <= 5.0) {
    f3 = f4 - 5.0;
  }

  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  /* 设置运行GPU参数定义 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

  ret = sscanf(nodename, "%*[^_]_%d", &igpu);
  if (ret == 1) {
    err = hipGetDeviceCount(&ngpu);
    if (err != hipSuccess) {
      fprintf(fplog, "===========================================\n");
      sprintf(str, "lsb_release -a     >>%s", logfilepath);
      if (system(str) != 0) {
        fprintf(stderr, "Command Executed Fail:%s\n", str);
        fprintf(fplog, "Command Executed Fail:%s\n", str);
        fflush(fplog);
      }

      sprintf(str, "lspci | grep 'VGA' >>%s", logfilepath);
      if (system(str) != 0) {
        fprintf(stderr, "Command Executed Fail:%s\n", str);
        fprintf(fplog, "Command Executed Fail:%s\n", str);
        fflush(fplog);
      }

      sprintf(str, "nvidia-smi         >>%s", logfilepath);
      if (system(str) != 0) {
        fprintf(stderr, "Command Executed Fail:%s\n", str);
        fprintf(fplog, "Command Executed Fail:%s\n", str);
        fflush(fplog);
      }

      sprintf(str, "free -g            >>%s", logfilepath);
      if (system(str) != 0) {
        fprintf(stderr, "Command Executed Fail:%s\n", str);
        fprintf(fplog, "Command Executed Fail:%s\n", str);
        fflush(fplog);
      }

      fprintf(fplog, "===========================================\n");

      fprintf(fplog, "Error: hipGetDevice Error %d\n", ngpu);
      fprintf(fplog, "       %s\n", hipGetErrorString(err));

      fflush(fplog);
    }
    if (igpu >= 0 && igpu < ngpu) {
      WARN("\n\nThis Machine has %d-GPU Card\n", ngpu);
    } else {
      igpu = igpu % ngpu;
      fprintf(stderr,
              "Warn: This Machine [%s] has no such GPU using id=%d instead\n",
              nodename, igpu);
      fprintf(fplog,
              "Warn: This Machine [%s] has no such GPU using id=%d instead\n",
              nodename, igpu);
    }
  } else {
    fprintf(stderr,
            "Error: Nodename [%s] Format Error or This Machine has no GPUs\n",
            nodename);
    fprintf(fplog,
            "Error: Nodename [%s] Format Error or This Machine has no GPUs\n",
            nodename);
    fflush(fplog);
    exit(1);
  }
  fflush(fplog);

  ngpu = 1;

  if (!getparint("ndata", &ndata)) {
    if (1 != readparfile(parfilepath, "ndata=%d", &ndata)) {
      ndata = ngpu;
    }
  }

  if (!getparint("ncons", &ncons)) {
    if (1 != readparfile(parfilepath, "ncons=%d", &ncons)) {
      ncons = ngpu;
    }
  }

  if (!getparint("poolsize", &poolsize)) {
    if (1 != readparfile(parfilepath, "poolsize=%d", &poolsize)) {
      poolsize = 10000;
    }
  }

  if (ndata > ngpu) ndata = ngpu;

  fprintf(stderr, "ndata=%d ncons=%d poolsize=%d\n", ndata, ncons, poolsize);
  fflush(fplog);

  /* 合理性检测以及变量内部化 */
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

  cdps = parseline(zcdps, &ncdps);

  if (zaspect[0] == 'A' || zaspect[0] == 'a') {
    aspect = 1;
  } else if (zaspect[0] == 'X' || zaspect[0] == 'x') {
    aspect = 2;
  } else if (zaspect[0] == 'Y' || zaspect[0] == 'y') {
    aspect = 3;
  } else if (zaspect[0] == 'N' || zaspect[0] == 'n') {
    aspect = 4;
  } else {
    aspect = 1;
  }

  ndipx = ((int)(dipx / ddipx + 0.5));
  ndipy = ((int)(dipy / ddipy + 0.5));

  nsmooth = 2;

  /* 获取成像空间参数 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  fci = cdp1;
  nxi = cdp2 - cdp1 + 1;
  iline = parseline(zline, &nyi);

  nzi = (int)(zmax / dzi + 0.5) + 1;

  /* 获取速度空间参数 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  if (!exist(velfilepath)) {
    fprintf(stderr, "Error: On Node [%s], Velocity File [%s] Does Not Exist\n",
            nodename, velfilepath);
    fprintf(fplog, "Error: On Node [%s], Velocity File [%s] Does Not Exist\n",
            nodename, velfilepath);
    fflush(fplog);
    exit(1);
  }
  fp = fopen(velfilepath, "rb");

  fgettr(fp, &tr);
  fseek(fp, -(240L + tr.ns * sizeof(float)), SEEK_END);
  fgettr(fp, &tr1);
  fclose(fp);

  scalev = getscale(tr.scalco);

  fxv = tr.gx * scalev;
  fyv = tr.gy * scalev;
  dxv = (tr1.gx - tr.gx) * scalev / (tr1.cdp - tr.cdp);
  if (tr.fldr == tr1.fldr) {
    if (!getparfloat("dyv", &dyv)) {
      if (1 != readparfile(parfilepath, "dyv=%f", &dyv)) {
        fprintf(stderr,
                "since first fldr and last fldr in the velocity file is the "
                "same:%d please specify dyv!\n",
                tr.fldr);
        fprintf(fplog,
                "since first fldr and last fldr in the velocity file is the "
                "same:%d please specify dyv!\n",
                tr.fldr);
        fflush(fplog);
        exit(1);
      }
    }
  } else {
    dyv = (tr1.gy - tr.gy) * scalev / (tr1.fldr - tr.fldr);
    /* Make Sure the Commandline is always Vaild */
    if (!getparfloat("dyv", &dyv)) {
      readparfile(parfilepath, "dyv=%f", &dyv);
    }
  }
  nxv = tr1.cdp - tr.cdp + 1;
  nyv = tr1.fldr - tr.fldr + 1;
  fcv = tr.cdp;
  flv = tr.fldr;
  ntv = tr.ns;
  dtv = tr.dt * 5.0E-7;
  dzv = 8;
  dzii = dzv * round(50 / dzv);
  fprintf(fplog, "dzv=%f dzii=%f\n", dzv, dzii);

  /* 获取数据空间参数 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  fld = line1;
  nyd = line2 - line1 + 1;

  /* 获取数据空间scalco->scaled */
  /* 合理性确认                 */
  line = iline[0];
  scaled = 1.0;
  for (i = line - 1000; i < line + 1000; ++i) {
    sprintf(str, "%s/off%d/line%d", offspath, 0, i);
    if (exist(str)) {
      fp = fopen(str, "rb");
      fgettr(fp, &tr);
      scaled = getscale(tr.scalco);

      if (tr.ns != ntl) {
        fprintf(stderr, "Error: In Data Space (tr.ns=%d)!=(ntl=%d)\n", tr.ns,
                ntl);
        fprintf(fplog, "Error: In Data Space (tr.ns=%d)!=(ntl=%d)\n", tr.ns,
                ntl);
        fflush(fplog);
        exit(1);
      }
      if (tr.dt != (int)(dt * 1.0E6)) {
        fprintf(stderr, "Error: In Data Space (tr.dt=%dms)!=(dt=%g*1.0E6)\n",
                tr.dt, dt);
        fprintf(fplog, "Error: In Data Space (tr.dt=%dms)!=(dt=%g*1.0E6)\n",
                tr.dt, dt);
        fflush(fplog);
        exit(1);
      }
      fclose(fp);
      break;
    }
  }

  /* 内部参数 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  dxi = dxv;
  dyi = dyv;
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  char nodename0[128];
  noffpath = alloc1char(1024);
  sscanf(nodename, "%[^_]s", nodename0);
  noffs = alloc1int(ndata);
  memset(noffs, 0, ndata * sizeof(int));

  for (idata = 0; idata < ndata; ++idata) {
    sprintf(offspath, "%s/data%d", projpath, idata);
    fprintf(stderr, "offspath=%s\n", offspath);
    sprintf(nodename, "%s_%d", nodename0, idata);
    fprintf(stderr, "nodename=%s\n", nodename);
    sprintf(noffpath, "%s/nofffile.rcd", offspath);

    if (!exist(noffpath)) {
      fprintf(stderr, "Error: On %s,[%s] Does Not Exist\n", nodename, noffpath);
      fprintf(fplog, "Error: On %s,[%s] Does Not Exist\n", nodename, noffpath);
      fflush(fplog);
      exit(1);
    }

    fp = fopen(noffpath, "r");
    if (fscanf(fp, "nofffile=%d\n", &noffs[idata]) != 1) {
      fprintf(stderr, "Read noffs from %s fail! in LINE:%d\n", noffpath,
              __LINE__);
      fprintf(fplog, "Read noffs from %s fail! in LINE:%d\n", noffpath,
              __LINE__);
      fflush(fplog);
      exit(1);
    }
    fclose(fp);
  }

  offs = alloc2float(noffs[0] * 2, ndata);
  memset(offs[0], 0, noffs[0] * 2 * ndata * sizeof(float));
  for (idata = 0; idata < ndata; ++idata) {
    sprintf(offspath, "%s/data%d", projpath, idata);
    sprintf(nodename, "%s_%d", nodename0, idata);
    for (ioffs = 0; ioffs < noffs[idata]; ++ioffs) {
      sprintf(str, "%s/off%d", offspath, ioffs);
      if (!exist(str)) {
        fprintf(stderr, "Error: On %s,[%s] Does Not Exist\n", nodename, str);
        fprintf(fplog, "Error: On %s,[%s] Does Not Exist\n", nodename, str);
        fflush(fplog);
        exit(1);
      }
      sprintf(str, "%s/off%d/ell_off%d", offspath, ioffs, ioffs);
      if (!exist(str)) {
        fprintf(stderr, "Error: On %s,[%s] Does Not Exist\n", nodename, str);
        fprintf(fplog, "Error: On %s,[%s] Does Not Exist\n", nodename, str);
        fflush(fplog);
        exit(1);
      }

      fp = fopen(str, "r");
      if (fscanf(fp, "%f,%f\n", &offmin, &offmax) != 2) {
        fprintf(stderr, "Read offmin and offmax from %s fail! in LINE:%d\n",
                str, __LINE__);
        fprintf(fplog, "Read offmin and offmax from %s fail! in LINE:%d\n", str,
                __LINE__);
        fflush(fplog);
        exit(1);
      }
      fclose(fp);

      offs[idata][ioffs] = offmin;
      offs[idata][ioffs + noffs[idata]] = offmax;
    }
  }

  /* 读取CIG(CRP)切除参数,并插值出偏移距组所对应的切除时间点 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  tomute = parsetuple(zmute, &ntomute, &npair);

  fprintf(fplog, "Mute Tuple:(DEPTH:m,OFFSET:m)\n");
  for (i = 0; i < ntomute; ++i) {
    fprintf(fplog, "%-5.2f %-5.0f\n", tomute[i * 2], tomute[i * 2 + 1]);
  }
  fprintf(fplog, "\n");

  tmute = alloc2float(noffs[0], ndata);
  memset(tmute[0], 0, noffs[0] * ndata * sizeof(float));

  time_offset_mute_santity(tomute, ntomute);

  for (idata = 0; idata < ndata; ++idata) {
    for (ioffs = 0; ioffs < noffs[idata]; ++ioffs) {
      tmute[idata][ioffs] = time_offset_mute(
          tomute, ntomute, offs[idata][ioffs]); /* offset-time pair */
      if (tmute[idata][ioffs] < 0) tmute[idata][ioffs] = 0;
      if (tmute[idata][ioffs] < zmin) tmute[idata][ioffs] = zmin;
    }
  }
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  sprintf(nodename, "%s_0", nodename0);
  fprintf(stderr, "%s - Start\n", nodename);
  fprintf(fplog, "%s - Start\n", nodename);
  fflush(fplog);

  time(&tic);

  qpsdm(ngpu, ndata, ncons, poolsize, projpath, projtask, dxi, nxi, dyi, nyi,
        nzi, dzi, dzii, zmin, fcv, fxv, flv, fyv, nxv, nyv, ntv, dtv, ndipx,
        ndipy, ddipx, ddipy, line1, line2, nyd, fld, ntl, dt, f1, fc, f3, f4,
        noffs, xline, iline, fci, iyi1, offs, cdps, ncdps, nsmooth, scaled,
        tmute, infopath, aspect, opcrp, outputpath, offspath, path, velfilepath,
        dipdirpath, f3dirpath, ttpath, nodename, threshold, contract, taperzone,
        nblock);

  time(&toc);

  fprintf(stderr, "%s,Time Elapse:%.0fs\n", nodename, difftime(toc, tic));
  fprintf(stderr, "%s:All Migration Work Done\n", nodename);

  fprintf(fplog, "%s,Time Elapse:%.0fs\n", nodename, difftime(toc, tic));
  fprintf(fplog, "%s:All Migration Work Done\n", nodename);
  fprintf(fplog, "\nTime: %s", ctime(&toc));
  fflush(fplog);

  // sprintf(str, "%s/break_point.txt", offspath);
  // unlink(str);

  // fprintf(fplog, "unlink BreakPoint:%s\n", str);
  // fflush(fplog);

  return 0;
}

void qpsdm(int ngpu, int ndata, int ncons, int poolsize, char *projpath,
           char *projtask, float dxi, int nxi, float dyi, int nyi, int nzi,
           float dzi, float dzii, float zmin, int fcv, double fxv, int flv,
           double fyv, int nxv, int nyv, int ntv, float dtv, int ndipx,
           int ndipy, float ddipx, float ddipy, int line1, int line2, int nyd,
           int fld, int ntl, float dt, float f1, float fc, float ff3, float f4,
           int *noffs, int extxline, int *iline, int fci, int iyi1,
           float **offs, int *cdps, int ncdps, int nsmooth, double scaled,
           float **tmute, char *infopath, int aspect, int opcrp,
           char *outputpath, char *offspath, char *path, char *velfilepath,
           char *dipdirpath, char *f3dirpath, char *ttpath, char *nodename,
           float threshold, float contract, float taperzone, int nblock) {
  (void)ngpu;
  (void)(nyd);
  (void)(fld);
  (void)(aspect);
  (void)(path);
  (void)(velfilepath);
  (void)(f3dirpath);
  (void)zmin;
  (void)ntv;
  (void)dtv;
  (void)f1;
  (void)nsmooth;
  (void)dipdirpath;

  char str[1024];

  float c1, c2;
  int i, i1;

  int ixi, iyi, iw;
  int ioffs;
  float odt1, df;
  double fxi, cyi;
  int n3, n4;

  int nfft, nffti;
  float dw, w;

  float *ww;
  int ioffl1, ioffl2;
  int line;
  FILE *fp;

  int cdp1;
  int izi;

  float **dipx1, **dipx2, **dipy1, **dipy2;
#ifdef QMIG3D
  float **f3;
  int **nf3;
#endif

  hipError_t err;

  char nodename0[128];

  time_t toc, tic;

  float **img = NULL, **tmpimg = NULL;

#ifdef QMIG3D
  float **Q;
#endif

  float dt1;

  int izmin;

  float tanx1 = tanf(60.0 / 180.0 * M_PI);
  float tanx2 = tanf(75.0 / 180.0 * M_PI);
  float tany1 = tanf(10.0 / 180.0 * M_PI);
  float tany2 = tanf(15.0 / 180.0 * M_PI);

  int ndipx2, ndipy2;

  int nxb1, ixstart, ixend, nxb;
  nxb1 = int(nxi / nblock + 0.5);

  /*aper*/
  int ****aper;
  int **aznx1, **azny1, **aznx2, **azny2;

  float eps;
  int naztab, *iaztab, i2, iaz;
  float c3;

  /* travle time talbe store with grid */
  /* in each grid the z is contigous   */
  int nxg, nzg, nxi1, nzi1;

  // int izibeg;
  // izibeg = zmin / dzi;
  fprintf(stderr, "nxi=%d, nzi=%d\n", nxi, nzi);

  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  eps = 0.001f;
  naztab = (int)(2.0 / eps + 1.5);
  iaztab = alloc1int(naztab);

  c2 = (1.0 - cosf(10.0 * 0.0174532925199)) * M_PI * 0.5; /* 方位角 */
  c2 = 1.0 - cosf(c2 * 0.5);
  i2 = (int)(c2 / eps);
  for (i = 0; i < i2; ++i) {
    iaztab[i] = 0;
  }
  for (iaz = 1; iaz < 18; ++iaz) {
    c2 = iaz * 10.0;
    c1 = c2 - 10.0;
    c3 = c2 + 10.0;

    c1 = (1.0 - cosf(c1 * 0.0174532925199)) * M_PI * 0.5; /* 方位角 */
    c2 = (1.0 - cosf(c2 * 0.0174532925199)) * M_PI * 0.5;
    c3 = (1.0 - cosf(c3 * 0.0174532925199)) * M_PI * 0.5;
    c1 = 1.0 - cosf(c2 - (c2 - c1) * 0.5);
    c2 = 1.0 - cosf(c2 + (c3 - c2) * 0.5);

    i1 = (int)(c1 / eps);
    i2 = (int)(c2 / eps);

    for (i = i1; i < i2; ++i) iaztab[i] = iaz;
  }
  for (i = i2; i < naztab; ++i) iaztab[i] = 0;

  /* 有效的孔径起算时间 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

  ndipx2 = ndipx * 2 + 1;
  ndipy2 = ndipy * 2 + 1;

  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  sscanf(nodename, "%[^_]s", nodename0);
  cdp1 = fci;

  /* 申请数据空间以及重采样空间 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  nfft = get_opt_n_fftw(ntl);
  nffti = get_opt_n_fftw(ntl * 2); /* 一般超过2倍的重采样率 */

  df = 1.0 / (nfft * dt);
  dw = 2.0 * M_PI * df;
  odt1 = nffti * df;
  dt1 = 1.0 / odt1;

  n3 = (int)(ff3 / df + 0.5);
  n4 = (int)(f4 / df + 0.5);

  ww = alloc1float(nffti / 2 + 1);

#if defined(HALF_DERIVATIVE) && defined(FULL_DERIVATIVE)
#error "Macro: HALF_DERIVATIVE & FULL_DERIVATIVE is both defined"
#elif !defined(HALF_DERIVATIVE) && !defined(FULL_DERIVATIVE)
#error "Macro: HALF_DERIVATIVE & FULL_DERIVATIVE is neither defined"
#endif

#ifdef HALF_DERIVATIVE
  for (iw = 0, w = 0.0; iw < n3; ++iw, w += dw) {
    ww[iw] = sqrtf(w);
  }
  c1 = n4 - n3;
  w = sqrtf(w);
  for (iw = n3; iw <= n4; ++iw) {
    c2 = cosf((iw - n3) / c1 * M_PI * 0.5);
    c2 *= c2;
    ww[iw] = w * c2;
  }
#elif defined(FULL_DERIVATIVE)
  for (iw = 0, w = 0.0; iw < n3; ++iw, w += dw) {
    ww[iw] = w;
  }
  c1 = n4 - n3;
  for (iw = n3; iw <= n4; ++iw) {
    c2 = cosf((iw - n3) / c1 * M_PI * 0.5);
    c2 *= c2;
    ww[iw] = w * c2;
  }
#endif
  for (iw = n4; iw < nffti / 2 + 1; ++iw) {
    ww[iw] = 0.0;
  }

  int nww = 110.0 / df;

  fprintf(stderr, "nww=%d\n", nww);
  fflush(fplog);

  /* 申请倾角场空间 */
  dipx1 = alloc2float(nzi, nxi);
  dipx2 = alloc2float(nzi, nxi);
  dipy1 = alloc2float(nzi, nxi);
  dipy2 = alloc2float(nzi, nxi);

#ifdef QMIG3D
  /* 申请nf3高通频空间 */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  f3 = alloc2float(nzi, nxi);
  nf3 = alloc2int(nzi, nxi);

  fprintf(fplog, "Create Space For F3\n");
  Q = alloc2float(nzi, nxi);
#endif

  /* 常量 */
  float lnGc, coef1c, coef2c, coef3c, coef4c;
  int ncoef1c;

  lnGc = logf(threshold);
  coef1c = threshold * (1.0 - lnGc - 2.5 * lnGc * lnGc);
  coef2c = threshold * (1.0 + 5.0 * lnGc);
  coef3c = -2.5 * threshold;
  coef4c = 1.1 * threshold;

  ncoef1c = (int)(10.0 / df);
  /* 倾角衰减模式系数拷贝到GPU */
  /*+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  contract *= 0.01745329251994;
  taperzone *= 0.01745329251994;

  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  /*++++++++++++++++++++++++    初始化GPU参数     ++++++++++++++++++++++*/
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/

  // 存储参数
  std::vector<GPUParamsNoLine *> gpuParamsNoLineVec;
  std::vector<GPUParamsWithLine *> gpuParamsWithLineVec;
  fprintf(fplog, "Initialize Line-Independent CPU/GPU Parameters Start ...\n");
  /*	初始化每个线程的GPU参数 */
  for (int igpu = 0; igpu < ncons; ++igpu) {
    hipSetDevice(igpu);

    GPUParamsNoLine *gpuParamsNoLine = new GPUParamsNoLine;
    GPUParamsWithLine *gpuParamsWithLine = new GPUParamsWithLine;

#ifdef QMIG3D
    float *re, *im;
    float *red, *imd;
    float *Qd;
    float *datav;
    fftwf_complex *wdatav;
    int *nf3d;
#endif

    int *itibeg;
    int *itibegd;

    float *dipx1d, *dipx2d, *dipy1d, *dipy2d;

    float *data = NULL;
    float *datad;

#ifdef QMIG3D
    re = alloc1float(ntl * 4);
    im = alloc1float(ntl * 4);
    hipMalloc(&red, sizeof(float) * ntl * 4);
    hipMalloc(&imd, sizeof(float) * ntl * 4);
    gpuParamsNoLine->re = re;
    gpuParamsNoLine->im = im;
    gpuParamsNoLine->red = red;
    gpuParamsNoLine->imd = imd;

    err = hipMalloc(&nf3d, sizeof(int) * nzi * nxi);
    if (err != hipSuccess) {
      fprintf(stderr, "Error: CUDA  Error in FILE:%s FUNCTION:%s LINE:%d\n",
              __FILE__, __FUNCTION__, __LINE__);
      fprintf(stderr, "       %s\n", hipGetErrorString(err));
      exit(1);
    }
    hipMalloc(&Qd, sizeof(float) * nxi * nzi);
    gpuParamsWithLine->nf3d = nf3d;
    gpuParamsWithLine->Qd = Qd;

    datav = fftwf_alloc_real(ntl * 4);
    wdatav = fftwf_alloc_complex(ntl * 4);
    gpuParamsWithLine->datav = datav;
    gpuParamsWithLine->wdatav = wdatav;
#endif

    itibeg = alloc1int(nxi);
    err = hipMalloc(&itibegd, sizeof(int) * nxi);
    if (err != hipSuccess) {
      fprintf(fplog, "Error: CUDA  Error in FILE:%s FUNCTION:%s LINE:%d\n",
              __FILE__, __FUNCTION__, __LINE__);
      fprintf(fplog, "       %s\n", hipGetErrorString(err));
      exit(1);
    }
    gpuParamsNoLine->itibeg = itibeg;
    gpuParamsNoLine->itibegd = itibegd;

    hipMalloc(&dipx1d, sizeof(float) * nxi * nzi);
    hipMalloc(&dipx2d, sizeof(float) * nxi * nzi);
    hipMalloc(&dipy1d, sizeof(float) * nxi * nzi);
    hipMalloc(&dipy2d, sizeof(float) * nxi * nzi);
    gpuParamsWithLine->dipx1d = dipx1d;
    gpuParamsWithLine->dipx2d = dipx2d;
    gpuParamsWithLine->dipy1d = dipy1d;
    gpuParamsWithLine->dipy2d = dipy2d;

    data = alloc1float(nffti);
    hipMalloc(&datad, sizeof(float) * nffti);
    gpuParamsWithLine->data = data;
    gpuParamsWithLine->datad = datad;

    gpuParamsNoLine->igpu = igpu;
    gpuParamsNoLineVec.push_back(gpuParamsNoLine);
    gpuParamsWithLineVec.push_back(gpuParamsWithLine);
  }

  /* 层速度模型空间 X方向两边各扩展1000m Y方向两边各扩展500m*/
  float fxv1, fyv1, exv1, eyv1;
  fxv1 = fxv - 2000;
  exv1 = fxv + nxv * dxi + 2000;
  fyv1 = fyv - 1000;
  eyv1 = fyv + nyv * dyi + 1000;

  int ttt_cdp1, ttt_cdp2;
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  /*+++++++++++++++++++++++++++    主循环     ++++++++++++++++++++++++*/
  /*++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++*/
  fprintf(fplog,
          "Initialize Line-dependent CPU/GPU Parameters And Start Migration "
          "With Multithreading ...\n");
  CPUParamsNoLine *cpuParamsNoLine = new CPUParamsNoLine;
  cpuParamsNoLine->dyi = dyi;
  cpuParamsNoLine->dxi = dxi;
  cpuParamsNoLine->nxi = nxi;
  cpuParamsNoLine->nzi = nzi;
  cpuParamsNoLine->ntl = ntl;
  cpuParamsNoLine->scaled = scaled;
  cpuParamsNoLine->fci = fci;
  cpuParamsNoLine->dzi = dzi;
  cpuParamsNoLine->ndipx = ndipx;
  cpuParamsNoLine->ndipy = ndipy;
  cpuParamsNoLine->ddipx = ddipx;
  cpuParamsNoLine->ddipy = ddipy;
  cpuParamsNoLine->dt = dt;
  cpuParamsNoLine->fc = fc;
  cpuParamsNoLine->eps = eps;
  cpuParamsNoLine->iaztab = iaztab;
  cpuParamsNoLine->fxv1 = fxv1;
  cpuParamsNoLine->fyv1 = fyv1;
  cpuParamsNoLine->exv1 = exv1;
  cpuParamsNoLine->eyv1 = eyv1;
  cpuParamsNoLine->nfft = nfft;
  cpuParamsNoLine->nffti = nffti;
  cpuParamsNoLine->ww = ww;
  cpuParamsNoLine->dt1 = dt1;
  cpuParamsNoLine->ndipx2 = ndipx2;
  cpuParamsNoLine->ndipy2 = ndipy2;
  cpuParamsNoLine->lnG = lnGc;
  cpuParamsNoLine->coef1 = coef1c;
  cpuParamsNoLine->coef2 = coef2c;
  cpuParamsNoLine->coef3 = coef3c;
  cpuParamsNoLine->coef4 = coef4c;
  cpuParamsNoLine->taperzone = taperzone;
  cpuParamsNoLine->dzii = dzii;
  /* 循环成像线 */
  for (iyi = iyi1; iyi < nyi; ++iyi) {
    time(&tic);

    line = iline[iyi];
    fxi = fxv + (fci - fcv) * dxi;
    cyi = fyv + (line - flv) * dyi;

    fprintf(fplog, "Start Image Line:%d\n", line);

    /* 确定一个偏移距文件夹下,对该成像线有贡献的数据域线范围 */
    ioffl1 = line - extxline;
    ioffl2 = line + extxline;
    if (ioffl1 < line1) ioffl1 = line1;
    if (ioffl2 > line2) ioffl2 = line2;

    /* store the original coor */
    coor_t *vel_coor = (coor_t *)malloc(sizeof(coor_t) * nxi);
    short vel_scalel;
    get_vel_coor(velfilepath, line, &vel_coor[0], &vel_scalel, fci, nxi, fcv,
                 nxv, flv, nyv);

    /* get travel time table */
    get_ttt_all(ttpath, "t", line, line1, line2, &nxg, &nzg, &nxi1, &nzi1,
                &ttt_cdp1, &ttt_cdp2, gpuParamsWithLineVec, ncons);
#ifdef QMIG3D
    int qqq_cdp1, qqq_cdp2;
    get_ttt_all(ttpath, "q", line, line1, line2, &nxg, &nzg, &nxi1, &nzi1,
                &qqq_cdp1, &qqq_cdp2, gpuParamsWithLineVec, ncons);
    if (ttt_cdp1 != qqq_cdp1 || ttt_cdp2 != qqq_cdp2 || ttt_cdp1 >= ttt_cdp2) {
      fprintf(
          fplog,
          "travel time table cdp range error, or not confirm with qfield\n");
      exit(1);
    }
#endif

    /* 将倾角转存为其tan值 */
    for (ixi = 0; ixi < nxi; ++ixi) {
      for (izi = 0; izi < nzi; ++izi) {
        dipx1[ixi][izi] =
            tanf(-10 * 0.0174532925199 + contract); /* *PI/180.0 */
        dipx2[ixi][izi] = tanf(10 * 0.0174532925199 - contract);
        dipy1[ixi][izi] = tanf(-10 * 0.0174532925199 + contract);
        dipy2[ixi][izi] = tanf(10 * 0.0174532925199 - contract);
      }
    }
#ifdef QMIG3D
    /* 读取高通频f3,插值 */
    for (ixi = 0; ixi < nxi; ++ixi) {
      for (izi = 0; izi < nzi; ++izi) {
        if (f3[ixi][izi] > ff3) {
          f3[ixi][izi] = 90;
        }

        nf3[ixi][izi] = (int)(f3[ixi][izi] / df + 0.5);
      }
    }
    fprintf(fplog, "Interpolation of F3 for line:%d\n", line);
#endif

#ifdef QMIG3D
    for (ixi = 0; ixi < nxi; ++ixi) {
      for (izi = 0; izi < nzi; ++izi) {
        Q[ixi][izi] = dw / (2.0 * (100.0 + izi * 100.0 / nzi) /*tr.data[izi]*/);
      }
    }
#endif

    for (int igpu = 0; igpu < ncons; ++igpu) {
      hipSetDevice(igpu);

      GPUParamsWithLine *gpuParamsWithLine = gpuParamsWithLineVec[igpu];

      hipMemcpy(gpuParamsWithLine->dipx1d, dipx1[0], sizeof(float) * nzi * nxi,
                 hipMemcpyHostToDevice);
      hipMemcpy(gpuParamsWithLine->dipx2d, dipx2[0], sizeof(float) * nzi * nxi,
                 hipMemcpyHostToDevice);
      hipMemcpy(gpuParamsWithLine->dipy1d, dipy1[0], sizeof(float) * nzi * nxi,
                 hipMemcpyHostToDevice);
      hipMemcpy(gpuParamsWithLine->dipy2d, dipy2[0], sizeof(float) * nzi * nxi,
                 hipMemcpyHostToDevice);

#ifdef QMIG3D
      hipMemcpy(gpuParamsWithLine->nf3d, nf3[0], sizeof(int) * nxi * nzi,
                 hipMemcpyHostToDevice);
      hipMemcpy(gpuParamsWithLine->Qd, Q[0], sizeof(float) * nxi * nzi,
                 hipMemcpyHostToDevice);
#endif
    }

    CPUParamsWithLine *cpuParamsWithLine = new CPUParamsWithLine;
    for (int iblock = 0; iblock < nblock; ++iblock) {
      ixstart = iblock * nxb1;
      ixend = (iblock + 1) * nxb1;
      if (iblock == nblock - 1) ixend = nxi;
      nxb = ixend - ixstart;
      cpuParamsWithLine->ixstart = ixstart;
      cpuParamsWithLine->ixend = ixend;
      cpuParamsWithLine->nxb = nxb;

      img = alloc2float(nzi, nxb);
      tmpimg = alloc2float(nzi, nxb);

      for (int igpu = 0; igpu < ncons; ++igpu) {
        hipSetDevice(igpu);

        float *imgd = NULL;

        hipMalloc(&imgd, sizeof(float) * nxb * nzi);
        gpuParamsWithLineVec[igpu]->imgd = imgd;
      }

      for (int idata = 0; idata < ndata; ++idata) {
        fprintf(fplog, "Create Apeture Azimuth Index Table\n");

        aznx1 = alloc2int(18, noffs[idata]);
        aznx2 = alloc2int(18, noffs[idata]);
        azny1 = alloc2int(18, noffs[idata]);
        azny2 = alloc2int(18, noffs[idata]);

        aper = NULL;
        fprintf(fplog, "Aperture For Line %d\n", line);
        fprintf(fplog, "Start Offset Group Loop\n");
        fflush(fplog);

        cpuParamsWithLine->aper = aper;
        cpuParamsWithLine->aznx1 = aznx1;
        cpuParamsWithLine->aznx2 = aznx2;
        cpuParamsWithLine->azny1 = azny1;
        cpuParamsWithLine->azny2 = azny2;

        sprintf(outputpath, "%s/%d/%s/result", projpath, idata, projtask);
        sprintf(nodename, "%s_%d", nodename0, idata);
        sprintf(offspath, "%s/data%d", projpath, idata);
        sprintf(infopath, "%s/%d/%s", projpath, idata, projtask);
        /* 循环偏移距组文件夹 */
        for (ioffs = 0; ioffs < noffs[idata]; ++ioffs) {
          fprintf(fplog, "Input Offset Group: %d of %d in Block%d \n",
                  ioffs + 1, noffs[idata], iblock);
          fflush(fplog);

          sprintf(str, "%s/%s.off%d.L%d.su", outputpath, nodename, ioffs, line);

          if (iblock == 0) {
            if (exist(str)) {
              fprintf(fplog, "File:%s Exist\n", str);
              if (0 == unlink(str))
                fprintf(fplog, "File:%s been unlinked\n", str);
              else
                fprintf(fplog, "File:%s unlink failure\n", str);
            }
          }

          /* 设置切除参数 */
          izmin = (int)(tmute[idata][ioffs] / dzi);

          if (izmin > nzi) {
            fprintf(
                fplog,
                "Warn: Offset=%f is greater than the offset mute parameter\n",
                tmute[idata][ioffs]);
            fprintf(fplog, "      No image is generate\n");
            fflush(fplog);
            continue;
          }

#ifdef QMIG3D
          /* according to offset mute -> split to two segment */
          float t00, t11;
          int it00pre = 0.48 / dt;
          int it00suf = 0.36 / dt;
          float tdstart1, tdstart2, tdmid;
          int itb1, itb2, nsb, nf1, nf4;
          int iti;

#if 0
          t00 = ts01[ioffs][0] - it00pre * dt;
          t00 = t00 > 0.0 ? t00 : 0.0;
          t11 = ts01[ioffs][1] + it00suf * dt;
          t11 = t11 < (ntl - 1) * dt ? t11 : (ntl - 1) * dt;
#endif
          t00 = 0;
          t11 = (ntl - 1) * dt;

          nsb = (t11 - t00 - it00pre * dt - it00suf * dt) / 2.0 / dt;

          itb1 = t00 / dt;
          tdstart1 = itb1 * dt;

          tdmid = t00 + it00pre * dt + nsb * dt;

          nsb = nsb + it00pre + it00suf;
          nsb = get_opt_n_fftw(nsb);

          itb2 = t11 / dt - nsb + 1;
          tdstart2 = itb2 * dt;

          fprintf(fplog, "offset =%f\n", offs[idata][ioffs]);
          fprintf(fplog, "t00 =%f\n", t00);
          fprintf(fplog, "t11 =%f\n", t11);
          fprintf(fplog, "tdstart1 =%f\n", tdstart1);
          fprintf(fplog, "tdmid    =%f\n", tdmid);
          fprintf(fplog, "tdstart2 =%f\n", tdstart2);
          fprintf(fplog, "nsb =%d\n", nsb);

          df = 1.0 / nsb / dt;
          dw = 2.0 * M_PI * df;
          nf1 = (int)(f1 / df);
          nf4 = (int)(f4 / df);
          nww = nf4 - nf1 + 1;

          ncoef1c = (int)(10.0 / df);

          for (ixi = 0; ixi < nxi; ++ixi) {
            for (iti = 0; iti < nzi; ++iti) {
              nf3[ixi][iti] = (int)(f3[ixi][iti] / df + 0.5);
            }
          }
#endif

          /* 初始化offset剖面 */
          if (opcrp) {
            memset(img[0], 0, sizeof(float) * nxb * nzi);
            memset(tmpimg[0], 0, sizeof(float) * nxb * nzi);
          }

          for (int igpu = 0; igpu < ncons; ++igpu) {
            hipSetDevice(igpu);

            GPUParamsWithLine *gpuParamsWithLine = gpuParamsWithLineVec[igpu];
            if (opcrp) {
              hipMemset(gpuParamsWithLine->imgd, 0, sizeof(float) * nxb * nzi);
            }

#ifdef QMIG3D
            gpuParamsWithLine->planv =
                fftwf_plan_dft_r2c_1d(nsb, gpuParamsWithLine->datav,
                                      gpuParamsWithLine->wdatav, FFTW_MEASURE);
            hipMemcpy(gpuParamsWithLine->nf3d, nf3[0], sizeof(int) * nxi * nzi,
                       hipMemcpyHostToDevice);
#endif
          }
          cpuParamsWithLine->fxi = fxi;
          cpuParamsWithLine->cyi = cyi;
          cpuParamsWithLine->dw = dw;
          cpuParamsWithLine->ioffs = ioffs;
#ifdef QMIG3D
          cpuParamsWithLine->itb1 = itb1;
          cpuParamsWithLine->itb2 = itb2;
          cpuParamsWithLine->nsb = nsb;
          cpuParamsWithLine->tdstart1 = tdstart1;
          cpuParamsWithLine->tdstart2 = tdstart2;
          cpuParamsWithLine->nf1 = nf1;
          cpuParamsWithLine->tdmid = tdmid;
          cpuParamsWithLine->qqq_cdp1 = qqq_cdp1;
          cpuParamsWithLine->qqq_cdp2 = qqq_cdp2;
#endif
          cpuParamsWithLine->nww = nww;
          cpuParamsWithLine->nxg = nxg;
          cpuParamsWithLine->nzg = nzg;
          cpuParamsWithLine->nxi1 = nxi1;
          cpuParamsWithLine->nzi1 = nzi1;
          cpuParamsWithLine->ttt_cdp1 = ttt_cdp1;
          cpuParamsWithLine->ttt_cdp2 = ttt_cdp2;
          cpuParamsWithLine->ncoef1 = ncoef1c;

          safe_queue<segy *> datapool(poolsize);
          pthread_t prod_thread;

          PROD_PARAMS *prod_params = new PROD_PARAMS;
          prod_params->datapool = &datapool;
          prod_params->ioffl1 = ioffl1;
          prod_params->ioffl2 = ioffl2;
          prod_params->offspath = offspath;
          prod_params->ioffs = ioffs;
          prod_params->ntl = ntl;
          prod_params->ncons = ncons;
          prod_params->nodename = nodename;

          if (pthread_create(&prod_thread, NULL, producer, prod_params) != 0) {
            fprintf(stderr, "Error: Producer Thread Create failed\n");
            exit(1);
          }

          pthread_t cons_threads[ncons];
          std::vector<CONS_PARAMS *> consParamVec;
          for (int igpu = 0; igpu < ncons; ++igpu) {
            GPUParamsNoLine *gpuParamsNoLine = gpuParamsNoLineVec[igpu];
            GPUParamsWithLine *gpuParamsWithLine = gpuParamsWithLineVec[igpu];

            /* consumer	*/
            CONS_PARAMS *cons_params = new CONS_PARAMS;
            cons_params->datapool = &datapool;
            cons_params->cpuParamsNoLine = cpuParamsNoLine;
            cons_params->cpuParamsWithLine = cpuParamsWithLine;
            cons_params->gpuParamsNoLine = gpuParamsNoLine;
            cons_params->gpuParamsWithLine = gpuParamsWithLine;

            if (pthread_create(&cons_threads[igpu], NULL, consumer,
                               cons_params) != 0) {
              fprintf(stderr, "Error: Consumer-%d Thread Create failed\n",
                      igpu);
              exit(1);
            }
            consParamVec.push_back(cons_params);
          }

          pthread_join(prod_thread, NULL);

          for (i = 0; i < ncons; ++i) {
            pthread_join(cons_threads[i], NULL);
          }

          delete prod_params;

          if (opcrp) {
            fprintf(fplog, "Get Image(crp) from GPU For Line %d\n", line);

            for (int igpu = 0; igpu < ncons; ++igpu) {
              hipSetDevice(igpu);

              GPUParamsWithLine *gpuParamsWithLine = gpuParamsWithLineVec[igpu];
              hipMemcpy(tmpimg[0], gpuParamsWithLine->imgd,
                         sizeof(float) * nxb * nzi, hipMemcpyDeviceToHost);

              for (i = 0; i < nxb * nzi; ++i) {
                img[0][i] += tmpimg[0][i];
              }
              memset(tmpimg[0], 0, sizeof(float) * nxb * nzi);
              delete consParamVec[igpu];
            }
            consParamVec.clear();

#ifdef QMIG3D
            for (ixi = 0; ixi < nxb; ++ixi) {
              for (iti = 0; iti < nzi; ++iti) {
                img[ixi][iti] /= nsb;
              }
            }
#endif
            fprintf(fplog, "Output:%s in Block%d \n", str, iblock);

            /* append binary */
            fp = fopen(str, "ab");

            if (fp != NULL) {
              memset(&tr, 0, sizeof(tr));
              tr.ns = nzi;
              tr.dt = (int)(dzi * 1.0E3);
              tr.fldr = line;
              tr.offset = (int)(offs[idata][ioffs]);
              tr.trid = 1;
              tr.counit = 1;
              /*  tr.delrt=(int)(itmin*dti*2.0E3); */
              tr.nvs = 1;
              i1 = 0;
              for (i = 0; i < ncdps; ++i) {
                ixi = cdps[i] - cdp1;
                if (ixi >= ixstart && ixi < ixend) {
                  ++i1;
                  tr.tracl = i1;
                  tr.cdp = fci + ixi;
                  /* output original coor in vel */
                  tr.sx = vel_coor[ixi].sx;
                  tr.sy = vel_coor[ixi].sy;
                  tr.gx = vel_coor[ixi].sx;
                  tr.gy = vel_coor[ixi].sy;
                  tr.scalco = vel_scalel;

                  memcpy((tr.data), img[ixi - ixstart], sizeof(float) * nzi);
                  fputtr(fp, &tr);
                }
              }
              fclose(fp);

              fprintf(fplog, "Write Image(crp) To Disk For Line %d:%s\n", line,
                      str);
              fflush(fplog);
            } else {
              fprintf(fplog, "Error: On [%s], File [%s] Open Error\n", nodename,
                      str);
              fprintf(fplog, "       %s\n", strerror(errno));

              exit(1);
            }
          }
        } /* 循环offs 结束 */
      }
    }

    delete cpuParamsWithLine;

    /* 输出局部检查点信息 */
    sprintf(str, "%s/break_point.txt", offspath);
    fp = fopen(str, "w");
    fprintf(fp, "%d\n", line);
    fclose(fp);

    sprintf(str, "%s/%s_%d", infopath, nodename, line);
    fp = fopen(str, "w");
    if (fp != NULL) {
      time(&toc);
      fprintf(fplog, "%s\n", ctime(&toc));
      fclose(fp);
    }

    for (int igpu = 0; igpu < ncons; ++igpu) {
      hipSetDevice(igpu);

      GPUParamsWithLine *gpuParamsWithLine = gpuParamsWithLineVec[igpu];
      reset_ttt_all(gpuParamsWithLine->ttt_descd, gpuParamsWithLine->tttd);
#ifdef QMIG3D
      reset_ttt_all(gpuParamsWithLine->qqq_descd, gpuParamsWithLine->qqqd);
#endif
    }

    time(&toc);
    fprintf(stderr, "Time Elapse of Line-%d:%.0fs\n", line, difftime(toc, tic));
  } /* 循环成像线 结束 */

  sprintf(infopath, "%s/%d/%s", projpath, 0, projtask);
  sprintf(nodename, "%s", nodename0);

  /* 输出任务完成信息(共享盘) */
  sprintf(str, "%s/%s", infopath, nodename);
  fp = fopen(str, "w");
  if (fp != NULL) {
    time(&toc);
    fprintf(fplog, "%s\n", ctime(&toc));
    fclose(fp);
  }
  /* 释放内存和显存 */
  /* ... */
}

void *producer(void *arg) {
  PROD_PARAMS *prod_params = (PROD_PARAMS *)arg;
  safe_queue<segy *> *datapool = prod_params->datapool;
  int ioffl1 = prod_params->ioffl1;
  int ioffl2 = prod_params->ioffl2;
  char *offspath = prod_params->offspath;
  int ioffs = prod_params->ioffs;
  int ntl = prod_params->ntl;
  int ncons = prod_params->ncons;
  char *nodename = prod_params->nodename;

  char str[1024];
  struct stat st;
  FILE *fp;
  int ntr;
  int itr;
  char nodename0[128];
  sscanf(nodename, "%[^_]s", nodename0);

  unsigned char *buf;
  int bufsize = (100 << 20); /* 100MB */

  buf = (unsigned char *)malloc(sizeof(unsigned char) * bufsize);

  for (int ioffl = ioffl1; ioffl <= ioffl2; ++ioffl) {
    sprintf(str, "%s/off%d/line%d", offspath, ioffs, ioffl);
    if (!exist(str)) continue; /* 如果文件不存在进行下一个 */

    /* 读取某一条数据域的有贡献线 进行成像 */
    stat(str, &st);
    if (st.st_size > bufsize) {
      bufsize = st.st_size;
      free(buf);
      buf = (unsigned char *)malloc(bufsize);
      WARN("Buffer Reallocate Triggered nodename=%s ioffs=%d ioffl=%d\n",
           nodename, ioffs, ioffl);
    }

    fp = fopen(str, "rb");
    if (fread(buf, 1L, st.st_size, fp) != (size_t)st.st_size) {
      fprintf(stderr, "Read %s fail:%s in LINE:%d\n", str, strerror(errno),
              __LINE__);
      fprintf(fplog, "Read %s fail:%s in LINE:%d\n", str, strerror(errno),
              __LINE__);
      fflush(fplog);
      exit(1);
    }
    fclose(fp);

    ntr = st.st_size / (240L + sizeof(float) * ntl);
    for (itr = 0; itr < ntr; ++itr) {
      segy *trace = new segy;
      swaptr(&buf[itr * (240L + sizeof(float) * ntl)], trace);
      datapool->push(trace);
    }
  }

  free(buf);
  segy *poison = NULL;
  for (int igpu = 0; igpu < ncons; ++igpu) {
    datapool->push(poison);
  }

  return NULL;
}

void *consumer(void *arg) {
  CONS_PARAMS *cons_params = (CONS_PARAMS *)arg;
  safe_queue<segy *> *datapool = cons_params->datapool;

  CPUParamsNoLine *cpuParamsNoLine = cons_params->cpuParamsNoLine;
  float dyi = cpuParamsNoLine->dyi;
  float dxi = cpuParamsNoLine->dxi;
  int nxi = cpuParamsNoLine->nxi;
  int nzi = cpuParamsNoLine->nzi;
  int ntl = cpuParamsNoLine->ntl;
  double scaled = cpuParamsNoLine->scaled;
  int fci = cpuParamsNoLine->fci;
  float dzi = cpuParamsNoLine->dzi;
  int ndipx = cpuParamsNoLine->ndipx;
  int ndipy = cpuParamsNoLine->ndipy;
  float ddipx = cpuParamsNoLine->ddipx;
  float ddipy = cpuParamsNoLine->ddipy;
  float dt = cpuParamsNoLine->dt;
  float fc = cpuParamsNoLine->fc;
  float eps = cpuParamsNoLine->eps;
  int *iaztab = cpuParamsNoLine->iaztab;
  float fxv1 = cpuParamsNoLine->fxv1;
  float fyv1 = cpuParamsNoLine->fyv1;
  float exv1 = cpuParamsNoLine->exv1;
  float eyv1 = cpuParamsNoLine->eyv1;
  int nfft = cpuParamsNoLine->nfft;
  int nffti = cpuParamsNoLine->nffti;
  float *ww = cpuParamsNoLine->ww;
  float dt1 = cpuParamsNoLine->dt1;
  int ndipx2 = cpuParamsNoLine->ndipx2;
  int ndipy2 = cpuParamsNoLine->ndipy2;
  float lnG = cpuParamsNoLine->lnG;
  float coef1 = cpuParamsNoLine->coef1;
  float coef2 = cpuParamsNoLine->coef2;
  float coef3 = cpuParamsNoLine->coef3;
  float coef4 = cpuParamsNoLine->coef4;
  float taperzone = cpuParamsNoLine->taperzone;
  float dzii = cpuParamsNoLine->dzii;

  CPUParamsWithLine *cpuParamsWithLine = cons_params->cpuParamsWithLine;
  int ****aper = cpuParamsWithLine->aper;
  int **aznx1 = cpuParamsWithLine->aznx1;
  int **aznx2 = cpuParamsWithLine->aznx2;
  int **azny1 = cpuParamsWithLine->azny1;
  int **azny2 = cpuParamsWithLine->azny2;
  double fxi = cpuParamsWithLine->fxi;
  double cyi = cpuParamsWithLine->cyi;
  float dw = cpuParamsWithLine->dw;
  int ioffs = cpuParamsWithLine->ioffs;
#ifdef QMIG3D
  int itb1 = cpuParamsWithLine->itb1;
  int itb2 = cpuParamsWithLine->itb2;
  int nsb = cpuParamsWithLine->nsb;
  float tdstart1 = cpuParamsWithLine->tdstart1;
  float tdstart2 = cpuParamsWithLine->tdstart2;
  int nf1 = cpuParamsWithLine->nf1;
  float tdmid = cpuParamsWithLine->tdmid;
#endif
  int nww = cpuParamsWithLine->nww;
  int nxg = cpuParamsWithLine->nxg;
  int nzg = cpuParamsWithLine->nzg;
  int nxi1 = cpuParamsWithLine->nxi1;
  int nzi1 = cpuParamsWithLine->nzi1;
  int ttt_cdp1 = cpuParamsWithLine->ttt_cdp1;
  int ttt_cdp2 = cpuParamsWithLine->ttt_cdp2;
  int qqq_cdp1 = cpuParamsWithLine->qqq_cdp1;
  int qqq_cdp2 = cpuParamsWithLine->qqq_cdp2;
  int ncoef1 = cpuParamsWithLine->ncoef1;
  int ixstart = cpuParamsWithLine->ixstart;
  int ixend = cpuParamsWithLine->ixend;
  int nxb = cpuParamsWithLine->nxb;

  GPUParamsNoLine *gpuParamsNoLine = cons_params->gpuParamsNoLine;
#ifdef QMIG3D
  float *re = gpuParamsNoLine->re;
  float *im = gpuParamsNoLine->im;
  float *red = gpuParamsNoLine->red;
  float *imd = gpuParamsNoLine->imd;
#endif
  int igpu = gpuParamsNoLine->igpu;
  int *itibeg = gpuParamsNoLine->itibeg;
  int *itibegd = gpuParamsNoLine->itibegd;

  GPUParamsWithLine *gpuParamsWithLine = cons_params->gpuParamsWithLine;
#ifdef QMIG3D
  int *nf3d = gpuParamsWithLine->nf3d;
  float *Qd = gpuParamsWithLine->Qd;
  float *datav = gpuParamsWithLine->datav;
  fftwf_complex *wdatav = gpuParamsWithLine->wdatav;
  fftwf_plan planv = gpuParamsWithLine->planv;
  ttt_desc_t *qqq_descd = gpuParamsWithLine->qqq_descd;
  float *qqqd = gpuParamsWithLine->qqqd;
#endif
  float *data = gpuParamsWithLine->data;
  float *datad = gpuParamsWithLine->datad;
  float *dipx1d = gpuParamsWithLine->dipx1d;
  float *dipx2d = gpuParamsWithLine->dipx2d;
  float *dipy1d = gpuParamsWithLine->dipy1d;
  float *dipy2d = gpuParamsWithLine->dipy2d;
  ttt_desc_t *ttt_descd = gpuParamsWithLine->ttt_descd;
  float *tttd = gpuParamsWithLine->tttd;
  float *imgd = gpuParamsWithLine->imgd;

  while (true) {
    segy *trace;
    datapool->pop(trace);
    if (trace == NULL) {
      break;
    }
    psdm_kernel(
        *trace, dyi, dxi, nxi, nzi, ntl, scaled, fci, dzi, ndipx, ndipy, ddipx,
        ddipy, dt, fc, eps, iaztab, fxv1, fyv1, exv1, eyv1, nfft, nffti, ww,
        dt1, ndipx2, ndipy2, taperzone, lnG, coef1, coef2, coef3, coef4, dzii,
        aper, aznx1, aznx2, azny1, azny2, fxi, cyi, dw, ioffs,
#ifdef QMIG3D
        itb1, itb2, nsb, tdstart1, tdstart2, nf1, tdmid,
#endif
        nww, nxg, nzg, nxi1, nzi1, ttt_cdp1, ttt_cdp2, qqq_cdp1, qqq_cdp2,
        ncoef1, ixstart, ixend, nxb,
#ifdef QMIG3D
        re, im, red, imd,
#endif
        igpu, itibeg, itibegd,
#ifdef QMIG3D
        nf3d, Qd, datav, wdatav, planv, qqq_descd, qqqd,
#endif
        data, datad, dipx1d, dipx2d, dipy1d, dipy2d, ttt_descd, tttd, imgd);
    delete trace;
  }

  fprintf(fplog, "GPU-%d Finish Migration. RETURN\n", igpu);

  return NULL;
}

void psdm_kernel(segy tr,
                 // CPUParamsNoLine
                 float dyi, float dxi, int nxi, int nzi, int ntl, double scaled,
                 int fci, float dzi, int ndipx, int ndipy, float ddipx,
                 float ddipy, float dt, float fc, float eps, int *iaztab,
                 float fxv1, float fyv1, float exv1, float eyv1, int nfft,
                 int nffti, float *ww, float dt1, int ndipx2, int ndipy2,
                 float taperzone, float lnG, float coef1, float coef2,
                 float coef3, float coef4, float dzii,
                 // CPUParamsWithLine
                 int ****aper, int **aznx1, int **aznx2, int **azny1,
                 int **azny2, double fxi, double cyi, float dw, int ioffs,
#ifdef QMIG3D
                 int itb1, int itb2, int nsb, float tdstart1, float tdstart2,
                 int nf1, float tdmid,
#endif
                 int nww, int nxg, int nzg, int nxi1, int nzi1, int ttt_cdp1,
                 int ttt_cdp2, int qqq_cdp1, int qqq_cdp2, int ncoef1,
                 int ixstart, int ixend, int nxb,
// GPUParamsNoLine
#ifdef QMIG3D
                 float *re, float *im, float *red, float *imd,
#endif
                 int igpu, int *itibeg, int *itibegd,
// GPUParamsWithLine
#ifdef QMIG3D
                 int *nf3d, float *Qd, float *datav, fftwf_complex *wdatav,
                 fftwf_plan planv, ttt_desc_t *qqq_descd, float *qqqd,
#endif
                 float *data, float *datad, float *dipx1d, float *dipx2d,
                 float *dipy1d, float *dipy2d, ttt_desc_t *ttt_descd,
                 float *tttd, float *imgd) {
  hipSetDevice(igpu);
  (void)nxi;
  (void)dt;
  (void)fc;
  (void)aper;
  (void)nww;
  (void)qqq_cdp1;
  (void)qqq_cdp2;
  (void)Qd;
  (void)ww;
  (void)nfft;

  double sx, sy, gx, gy, ddx, ddy, offset;
  int cdpx, cdpy;
  float cosa, cosx, c1;
  int i, iaz, izimin, ixi, ixi1, ixi2, iti;
  int ntibeg, naznx, naznx1, naznx2, nazny1, nazny2;

  hipError_t err;

  sx = tr.sx * scaled;
  sy = tr.sy * scaled;
  gx = tr.gx * scaled;
  gy = tr.gy * scaled;

  if (sx < fxv1 || sx > exv1) return;
  if (gx < fxv1 || gx > exv1) return;
  if (sy < fyv1 || sy > eyv1) return;
  if (gy < fyv1 || gy > eyv1) return;

  ddx = gx - sx;
  ddy = gy - sy;

  offset = sqrtf(ddx * ddx + ddy * ddy);

  cdpx = (int)(((sx + gx) * 0.5 - fxi) / dxi + 0.5);
  cdpy = (int)(((sy + gy) * 0.5 - cyi) / dyi + 0.5);

  /* 由坐标求取其对应的方位角 */
  if (offset > 1.0) {
    cosa = ddx / offset;
    cosx = fabs(cosa);

    if (ddx * ddy < 0.0) cosx = -cosx;
    c1 = 1.0 - cosx;
    i = (int)(c1 / eps);
    iaz = iaztab[i];
  } else {
    iaz = 0;
  }

  /* 获取孔径参数 */
  naznx1 = aznx1[ioffs][iaz];
  naznx2 = aznx2[ioffs][iaz];
  nazny1 = azny1[ioffs][iaz];
  nazny2 = azny2[ioffs][iaz];
  naznx = naznx1 + naznx2;

  if (cdpy > nazny1 || cdpy < -nazny2) return;

  izimin = nzi;
  for (ixi = 0, ntibeg = 0; ixi < naznx; ++ixi) {
    iti = 0;
    if (iti < nzi - 10 && cdpx - naznx1 + ixi >= ixstart &&
        cdpx - naznx1 + ixi < ixend) {
      if (iti < izimin) {
        izimin = iti;
      }
      itibeg[ntibeg] = iti;
      ixi2 = cdpx - naznx1 + ixi;
      ++ntibeg;
    }
  }

  ixi1 = ixi2 - ntibeg + 1;

  if (ntibeg == 0) return;

#ifdef QMIG3D
  int iw;
  float w;
  for (int it = 0; it < ntl; ++it) {
    if (it * dt < 0.1)
      tr.data[it] *= 0.1 * 0.1;
    else
      tr.data[it] *= (it * dt) * (it * dt);
  }
  fetch_and_taper(tr.data, ntl, 0, ntl - 1, data);
  fetch_and_taper(data, ntl, itb1, itb1 + nsb - 1, datav);
  fftwf_execute(planv); /* tdata->wdata */

  if (is3D) /* 3-Dimension: Full Derivate */
  {
    for (iw = 0, w = nf1 * dw; iw < nww; ++iw, w += dw) {
      re[iw] = +wdatav[iw + nf1][1] * w;
      im[iw] = -wdatav[iw + nf1][0] * w;
    }
  } else /* 2-Dimension: Half Derivate */
  {
    for (iw = 0, w = nf1 * dw; iw < nww; ++iw, w += dw) {
      float cc = sqrtf(w) / sqrtf(2.0);
      re[iw] = cc * (wdatav[iw + nf1][0] + wdatav[iw + nf1][1]);
      im[iw] = cc * (wdatav[iw + nf1][1] - wdatav[iw + nf1][0]);
    }
  }

  fetch_and_taper(data, ntl, itb2, itb2 + nsb - 1, datav);
  fftwf_execute(planv); /* tdata->wdata */

  if (is3D) /* 3-Dimension: derivate */
  {
    for (iw = 0, w = nf1 * dw; iw < nww; ++iw, w += dw) {
      re[iw + nww] = +wdatav[iw + nf1][1] * w;
      im[iw + nww] = -wdatav[iw + nf1][0] * w;
    }
  } else /* 2-Dimension: derivate */
  {
    for (iw = 0, w = nf1 * dw; iw < nww; ++iw, w += dw) {
      float cc = sqrtf(w) / sqrtf(2.0);
      re[iw + nww] = cc * (wdatav[iw + nf1][1] + wdatav[iw + nf1][1]);
    }
  }

  hipMemcpy(red, re, sizeof(float) * 2 * nww, hipMemcpyHostToDevice);
  hipMemcpy(imd, im, sizeof(float) * 2 * nww, hipMemcpyHostToDevice);
#endif
  sx -= fxi;
  gx -= fxi;
  sy -= cyi;
  gy -= cyi;

  hipMemcpy(itibegd, itibeg, sizeof(int) * ntibeg, hipMemcpyHostToDevice);

  dim3 block(32, 16);
  dim3 grid((int)(ceil((nzi - izimin) / 32.0)), (int)(ceil((ntibeg) / 16.0)));

  image_depth_gpu<<<grid, block, nww * 4 * sizeof(float)>>>(
      imgd, izimin, nzi, datad, nffti, dt1, ixstart, ixend, nxb, dzii, red, imd,
      nf1, fc, dw, nww, nf3d, qqq_descd, qqqd, tdstart1, tdstart2, tdmid,
      taperzone, lnG, coef1, coef2, coef3, coef4, ncoef1, nxg, nzg, nxi1, nzi1,
      ttt_cdp1 - fci, ttt_cdp2 - fci, sx, sy, gx, gy, dxi, dzi, itibegd, ixi1,
      ixi2, ndipx, ndipx2, ddipx, ndipy, ndipy2, ddipy, ttt_descd, tttd, dipx1d,
      dipx2d, dipy1d, dipy2d);
  if ((err = hipGetLastError()) != hipSuccess) {
    fprintf(fplog, "Error: CUDA  Error in FILE:%s FUNCTION:%s LINE:%d\n",
            __FILE__, __FUNCTION__, __LINE__);
    fprintf(fplog, "       %s\n", hipGetErrorString(err));
    exit(1);
  }
}

__global__ void image_depth_gpu(
    float *img, int izimin, int nzi, float *data, int nt, float dt, int ixstart,
    int ixend, int nxb, float dzii,
#ifdef QMIG3D
    const float *__restrict__ red, const float *__restrict__ imd, int nf1,
    float fc, float dw, int nww, int *nf3, ttt_desc_t *qqq_desc, float *qqq,
    float tstart1, float tstart2, float tmid,
#endif
    float taperzone, float lnG, float coef1, float coef2, float coef3,
    float coef4, int ncoef1, int nxg, int nzg, int nxi1, int nzi1, int ttt_cdp1,
    int ttt_cdp2, float sx, float sy, float gx, float gy, float dxi, float dzi,
    int *itibegd, int ixi1, int ixi2, int ndipx, int ndipx2, float ddipx,
    int ndipy, int ndipy2, float ddipy, ttt_desc_t *ttt_desc, float *ttt,
    float *dipx1, float *dipx2, float *dipy1, float *dipy2) {
  int izi = blockIdx.x * blockDim.x + threadIdx.x + izimin;
  int ix = blockIdx.y * blockDim.y + threadIdx.y;

  float ts, tg, t;
  int it1, it2;

  float x, y, z, dsx, dgx;
  float tanx, tany, tanx1, tanx2, tany1, tany2, tanx1c, tanx2c, tany1c, tany2c;

  float w, tr;

  float c1, c2;
  float ds, dg;
  float v;
  float z2;

  int ixg, izg;
  float tst;

#ifdef QMIG3D
  extern __shared__ float shared[];
  float *re, *im;

  re = &shared[0];
  im = &shared[nww * 2];
  for (int i = threadIdx.y * blockDim.x + threadIdx.x; i < nww * 2;
       i += blockDim.x * blockDim.y) {
    re[i] = red[i];
    im[i] = imd[i];
  }

  __syncthreads();
#endif

  int izii, izii1, izg1;

  int ixi = ixi1 + ix;

  if (ix >= nxb) return;
  if (izi >= nzi || izi < itibegd[ix] || ixi > ixi2 || ixi < ttt_cdp1 ||
      ixi > ttt_cdp2)
    return;

  int ixzi = ixi * nzi + izi;

  z = izi * dzi;
  z2 = z * z;
  x = ixi * dxi;

  dsx = sx - x;
  dgx = gx - x;

  /* dip */
  ds = sqrtf(dsx * dsx + sy * sy + z2);
  dg = sqrtf(dgx * dgx + gy * gy + z2);
  x = dsx / ds + dgx / dg;
  y = sy / ds + gy / dg;
  z = z / ds + z / dg;

  tanx = x / z;
  tany = y / z;

  tanx1 = dipx1[ixzi];
  tanx1c = tanx1 - taperzone * (1.0f + tanx1 * tanx1);
  tanx2 = dipx2[ixzi];
  tanx2c = tanx2 + taperzone * (1.0f + tanx2 * tanx2);

  tany1 = dipy1[ixzi];
  tany1c = tany1 - taperzone * (1.0f + tany1 * tany1);
  tany2 = dipy2[ixzi];
  tany2c = tany2 + taperzone * (1.0f + tany2 * tany2);

  if (tanx <= tanx1c || tanx2c <= tanx || tany <= tany1c || tany2c <= tany)
    return;

  w = 1.0f;

  if (tanx < tanx1) w *= (tanx - tanx1c) / (tanx1 - tanx1c);
  if (tanx > tanx2) w *= (tanx2c - tanx) / (tanx2c - tanx2);
  if (tany < tany1) w *= (tany - tany1c) / (tany1 - tany1c);
  if (tany > tany2) w *= (tany2c - tany) / (tany2c - tany2);

  /* find the grid index */
  z = izi * dzi;
  ixg = (ixi - ttt_cdp1) / nxi1;
  izii = (int)z / dzii;
  izg = izii / nzi1;

  c2 = z / dzii - izii;
  c1 = 1 - c2;
  ts = 1.0;
  tg = 1.0;
  t = ts + tg;

  it1 = t / dt;
  it2 = it1 + 1;

  if (it1 > 0 && it2 < nt) {
#ifdef QMIG3D
    t = ts + tg;

    if (t > tmid) {
      re = &shared[nww];
      im = &shared[3 * nww];
      tst = t - tstart2;
    }
    v = 100;
    if (t > 0.1)
      v /= t * t;
    else
      v *= 100.0f;
#endif
    tr = (ds / dg * ds / dg * ds / dg * tg / ts);
    tr = w * (tr + 1.0 / tr);
    v *= tr;

    ixi -= ixstart;
    ixzi = ixi * nzi + izi;
    img[ixzi] += v;
  }
}

int get_ttt_all(const char *ttpath, const char *torq, int line, int line1,
                int line2, int *nxg, int *nzg, int *nxi1, int *nzi1,
                int *ttt_cdp1, int *ttt_cdp2,
                std::vector<GPUParamsWithLine *> gpuParamsWithLineVec,
                int ncons) {
  char index_file[4096], data_file[4096];
  char index_suff[10], data_suff[10];
  ttt_desc_t *ttt_desc;
  FILE *fp;
  int nxg0, nzg0, nxi0, nzi0;
  ssize_t ret;

  float *ttt = NULL;
  float *ttt1;
  size_t size = 0;

  int cdp1, cdp2;
  float dx, dz;

  // choose ttt or qqq
  if (strcmp(torq, "t") == 0) {
    sprintf(index_suff, "tidx");
    sprintf(data_suff, "ttt");
  } else {
    sprintf(index_suff, "qidx");
    sprintf(data_suff, "qqq");
  }

  // get index
  // =======================================================
  sprintf(index_file, "%s/Lxxx.%s", ttpath, index_suff);
  printf("load index file: %s\n", index_file);
  fp = fopen(index_file, "rb");

  if (fread(&cdp1, sizeof(int), 1, fp) != 1) {
    fprintf(stderr, "Error: Read cdp1 in %s\n", index_file);
  };
  if (fread(&cdp2, sizeof(int), 1, fp) != 1) {
    fprintf(stderr, "Error: Read cdp2 in %s\n", index_file);
  };
  if (fread(&dx, sizeof(float), 1, fp) != 1) {
    fprintf(stderr, "Error: Read dx in %s\n", index_file);
  };
  if (fread(&dz, sizeof(float), 1, fp) != 1) {
    fprintf(stderr, "Error: Read dz in %s\n", index_file);
  };
  if (fread(&nzg0, sizeof(int), 1, fp) != 1) {
    fprintf(stderr, "Error: Read nzg0 in %s\n", index_file);
  };
  if (fread(&nxg0, sizeof(int), 1, fp) != 1) {
    fprintf(stderr, "Error: Read nxg0 in %s\n", index_file);
  };
  if (fread(&nzi0, sizeof(int), 1, fp) != 1) {
    fprintf(stderr, "Error: Read nzi0 in %s\n", index_file);
  };
  if (fread(&nxi0, sizeof(int), 1, fp) != 1) {
    fprintf(stderr, "Error: Read nxi0 in %s\n", index_file);
  };

  *nxg = nxg0;
  *nzg = nzg0;
  *nxi1 = nxi0;
  *nzi1 = nzi0;

  *ttt_cdp1 = cdp1;
  *ttt_cdp2 = cdp2;

  printf("line = %d, nzg=%d, nxg=%d, nzi0=%d, nxi0=%d\n", line, nzg0, nxg0,
         nzi0, nxi0);
  printf("ttt_cdp1 = %d, ttt_cdp2 = %d\n", cdp1, cdp2);

  ttt_desc = (ttt_desc_t *)malloc(sizeof(ttt_desc_t) * nxg0 * nzg0);
  ret = fread(ttt_desc, sizeof(ttt_desc_t), nxg0 * nzg0, fp);
  fclose(fp);

  if (ret != (ssize_t)(nxg0 * nzg0)) {
    fprintf(stderr, " travle time table or qfield index size error %s\n",
            index_file);
    exit(1);
  }

  for (int igpu = 0; igpu < ncons; ++igpu) {
    hipSetDevice(igpu);

    GPUParamsWithLine *gpuParamsWithLine = gpuParamsWithLineVec[igpu];

    if (strcmp(torq, "t") == 0) {
      ttt_desc_t *ttt_descd;
      hipMalloc(&ttt_descd, sizeof(ttt_desc_t) * nxg0 * nzg0);
      hipMemcpy(ttt_descd, ttt_desc, sizeof(ttt_desc_t) * nxg0 * nzg0,
                 hipMemcpyHostToDevice);
      gpuParamsWithLine->ttt_descd = ttt_descd;
    } else {
      ttt_desc_t *qqq_descd;
      hipMalloc(&qqq_descd, sizeof(ttt_desc_t) * nxg0 * nzg0);
      hipMemcpy(qqq_descd, ttt_desc, sizeof(ttt_desc_t) * nxg0 * nzg0,
                 hipMemcpyHostToDevice);
      gpuParamsWithLine->qqq_descd = qqq_descd;
    }
  }

  free(ttt_desc);

  // load travel time table bin/or qfield
  // ========================================================

  int line1find = -1, line2find = -1;
  // find line -> line1
  for (int iline = line; iline >= line1; --iline) {
    sprintf(data_file, "%s/L%d.%s", ttpath, iline, data_suff);

    if (fexist(data_file)) {
      line1find = iline;
      break;
    }
  }

  // find line -> line2
  for (int iline = line; iline <= line2; ++iline) {
    sprintf(data_file, "%s/L%d.%s", ttpath, iline, data_suff);

    if (fexist(data_file)) {
      line2find = iline;
      break;
    }
  }

  printf("%d -> %d -> %d\n", line1find, line, line2find);

  fprintf(stderr, "load travel time table success: line %d\n", line);

  for (int igpu = 0; igpu < ncons; ++igpu) {
    hipSetDevice(igpu);

    GPUParamsWithLine *gpuParamsWithLine = gpuParamsWithLineVec[igpu];

    if (strcmp(torq, "t") == 0) {
      float *tttd;
      hipMalloc(&tttd, size);
      hipMemcpy(tttd, ttt, size, hipMemcpyHostToDevice);
      gpuParamsWithLine->tttd = tttd;
    } else {
      float *qqqd;
      hipMalloc(&qqqd, size);
      hipMemcpy(qqqd, ttt, size, hipMemcpyHostToDevice);
      gpuParamsWithLine->qqqd = qqqd;
    }
  }

  if (line1find == line && line2find == line) {
    // load travel time table line1find only
    sprintf(data_file, "%s/L%d.%s", ttpath, line, data_suff);
    printf("data_file = %s\n", data_file);

    size = fsize(data_file);
    ttt = (float *)malloc(size);

    fp = fopen(data_file, "rb");
    ret = fread(ttt, sizeof(char), size, fp);
    fclose(fp);

    if (ret != (ssize_t)size) {
      fprintf(stderr, " travle time table size error\n");
      exit(1);
    }
  } else if (line1find != -1 && line2find != -1) {
    // load travel time table line1find
    sprintf(data_file, "%s/L%d.%s", ttpath, line1find, data_suff);
    printf("data_file = %s\n", data_file);

    size = fsize(data_file);
    ttt = (float *)malloc(size);

    fp = fopen(data_file, "rb");
    ret = fread(ttt, sizeof(char), size, fp);
    fclose(fp);

    if (ret != (ssize_t)size) {
      fprintf(stderr, " travle time table size error\n");
      exit(1);
    }

    // load travel time table line2find
    sprintf(data_file, "%s/L%d.%s", ttpath, line2find, data_suff);
    printf("data_file = %s\n", data_file);

    if (size != fsize(data_file)) {
      printf("travel time table size not confirm\n");
    }
    ttt1 = (float *)malloc(size);

    fp = fopen(data_file, "rb");
    ret = fread(ttt1, sizeof(char), size, fp);
    fclose(fp);

    if (ret != (ssize_t)size) {
      fprintf(stderr, " travle time table size error\n");
      exit(1);
    }

    float c1, c2, t, t1;

    c2 = 1.0 * (line - line1find) / (line2find - line1find);
    c1 = 1.0 - c2;

    for (size_t i = 0; i < size / 4; ++i) {
      t = ttt[i];
      t1 = ttt1[i];

      t = t * c1 + t1 * c2;

      ttt[i] = t;
    }

    free(ttt1);
  } else {
    printf("no travel time table exists\n");
    exit(1);
  }

  fprintf(stderr, "load travel time table success: line %d\n", line);

  for (int igpu = 0; igpu < ncons; ++igpu) {
    hipSetDevice(igpu);

    GPUParamsWithLine *gpuParamsWithLine = gpuParamsWithLineVec[igpu];

    if (strcmp(torq, "t") == 0) {
      hipMalloc(&(gpuParamsWithLine->tttd), size);
      hipMemcpy(gpuParamsWithLine->tttd, ttt, size, hipMemcpyHostToDevice);
    } else {
      hipMalloc(&(gpuParamsWithLine->qqqd), size);
      hipMemcpy(gpuParamsWithLine->qqqd, ttt, size, hipMemcpyHostToDevice);
    }
  }

  free(ttt);

  return 0;
  // ========================================================
}

void reset_ttt_all(ttt_desc_t *desc_out, float *data_out) {
  hipFree(desc_out);
  hipFree(data_out);
}
